#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
#include <hip/hip_fp16.h>
#define MIN_VALUE (-1e38)







#define MM8_ONE_JSPLIT 16
#define MM8_ONE_TILE 1024



__global__ void kernel_mm8_one(
    const int N, const int M,
    const float *__restrict__ const x,
    const uint8_t *__restrict__ const w, const int w_stride,
    float *__restrict__ const y,
    const float *__restrict__ const r
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        float y_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j])
                
            );
        }
        atomicAdd(reinterpret_cast<float *>(&y[k]), *reinterpret_cast<float *>(&y_local));
    }
}



__global__ void kernel_mm8_three(
    const int N, const int M,
    const float *__restrict__ const x,
    const float *__restrict__ const x1,
    const float *__restrict__ const x2,

    const uint8_t *__restrict__ const w, const int w_stride,
    const uint8_t *__restrict__ const w1, const int w1_stride,
    const uint8_t *__restrict__ const w2, const int w2_stride,
    float *__restrict__ const y,
    float *__restrict__ const y1,
    float *__restrict__ const y2,
    const float *__restrict__ const r,
    const float *__restrict__ const r1,
    const float *__restrict__ const r2
    
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        float y_local = 0;
        float y1_local = 0;
        float y2_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j]));
            y1_local += x1[j] * (
                (w1[j * w1_stride + k] * r1[j]));
            y2_local += x2[j] * (
                (w2[j * w2_stride + k] * r2[j]));
           
        }
        atomicAdd(reinterpret_cast<float *>(&y[k]), *reinterpret_cast<float *>(&y_local));
        atomicAdd(reinterpret_cast<float *>(&y1[k]), *reinterpret_cast<float *>(&y1_local));
        atomicAdd(reinterpret_cast<float *>(&y2[k]), *reinterpret_cast<float *>(&y2_local));
    }
}
// generic T either float or fp16 or fp64

void cuda_mm8_three(int N, int M,
                    float *x,
                    float *x1,
                    float *x2,
                    uint8_t *w, int w_stride,
                    uint8_t *w1, int w1_stride,
                    uint8_t *w2, int w2_stride,
                    float *y,
                    float *y1,
                    float *y2,
                    float *r  ,
                    float *r1,
                    float *r2 
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_three<<<gridSize, blockSize>>>(
        N, M, x, x1, x2, w, w_stride, w1, w1_stride, w2, w2_stride, y, y1, y2, r, r1, r2);
}
void cuda_mm8_one(int N, int M,
                  float *x,
                  uint8_t *w, int w_stride,
                  float *y,
                  float *r   
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_one<<<gridSize, blockSize>>>(
        N, M, x, w, w_stride,y, r);
                }
